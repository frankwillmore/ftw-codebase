#include "hip/hip_runtime.h"
/* gfg2fvi256.cu */

#include <stdlib.h>
#include <math.h>
#include <stdio.h>

extern "C" {
#include <ftw_param.h>
#include <ftw_config_parser.h>
#include <ftw_types.h>
}

ftw_EnergyArray256 *GFGToRepulsion256_612(ftw_GFG16M *gfg, float sigma, float epsilon);

//  IN:    A pointer to a ***non-replicated*** polymer configuration as *ftw_GFG16M.  
//  OUT:   A free volume intensity is returned as *ftw_FVI256.  
//  Input configuration is not modified.  
//  Cross-interaction values are stored for the replicated config only.

__global__ void EnergyKernel256_612(ftw_GFG16M *d_configuration, ftw_EnergyArray256 *d_attraction, ftw_EnergyArray256 *d_repulsion, ftw_EnergyArray256 *d_total) {
  unsigned int idx = blockIdx.x;
  unsigned int idy = blockIdx.y;
  unsigned int idz = threadIdx.x;

  float repulsion=0;
  float attraction=0;
  float sigma_over_r_sq;
  float dx, dy, dz, dd;
  float f_resolution_x = d_configuration->box_x / 256;
  float f_resolution_y = d_configuration->box_y / 256;
  float f_resolution_z = d_configuration->box_z / 256;

  float cuda_x = idx * f_resolution_x;
  float cuda_y = idy * f_resolution_y;
  float cuda_z = idz * f_resolution_z;

  // evaluate energy at (cuda_x, cuda_y, cuda_z);
  for (int i=0; i< d_configuration->n_atoms; i++) {
    // central atom
    dx = d_configuration->atom[i].x - cuda_x;
    dy = d_configuration->atom[i].y - cuda_y;
    dz = d_configuration->atom[i].z - cuda_z;
    dd = dx*dx + dy*dy + dz*dz; 
    sigma_over_r_sq = d_configuration->atom[i].sigma * d_configuration->atom[i].sigma / dd; // squared   
    repulsion += d_configuration->atom[i].epsilon * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq;
    attraction += d_configuration->atom[i].epsilon * sigma_over_r_sq * sigma_over_r_sq * sigma_over_r_sq;
  } 

  // If NULL pointers are passed for the attraction or repulsion, no values are returned.
  if (d_attraction) d_attraction->energy[idx][idy][idz] = 4 * attraction;
  if (d_repulsion) d_repulsion->energy[idx][idy][idz] = 4 * repulsion;
  if (d_total) d_total->energy[idx][idy][idz] = 4 * repulsion - 4 * attraction;
}

ftw_EnergyArray256 *GFGToRepulsion256_612(ftw_GFG16M *gfg, float sigma, float epsilon)
{
  ftw_EnergyArray256 	*d_repulsion;
  ftw_GFG16M 		*d_configuration;

  // replicate the gfg
  ftw_GFG16M *h_configuration = replicateGFG16M(gfg); 

// and cross-parameterize use 6-12 rule
  for (int n=0; n<gfg->n_atoms; n++) {
    h_configuration->atom[n].sigma = 0.5f * (sigma + h_configuration->atom[n].sigma);
    h_configuration->atom[n].epsilon = sqrt(epsilon * h_configuration->atom[n].epsilon);
  }

  // then do the calc
  hipError_t err;
  /* allocate for energy array and configuration on device */
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_repulsion, sizeof(ftw_EnergyArray256)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMalloc( (void **) &d_configuration, sizeof(ftw_GFG16M)));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy( d_configuration, h_configuration, sizeof(ftw_GFG16M), hipMemcpyHostToDevice ));

  dim3 dimGrid(256, 256);
  dim3 dimBlock(256, 1, 1);

  EnergyKernel256_612<<< dimGrid, dimBlock >>>(d_configuration, NULL, d_repulsion, NULL);
  hipDeviceSynchronize(); // block until the device has completed
  err = hipGetLastError();
  if (err != hipSuccess) printf("%s\n", hipGetErrorString(err)); 

  // retrieve result
  ftw_EnergyArray256 *h_repulsion = (ftw_EnergyArray256 *)malloc(sizeof(ftw_EnergyArray256));
  for(err = hipErrorUnknown; err != hipSuccess; err = hipMemcpy(h_repulsion, d_repulsion, sizeof(ftw_EnergyArray256), hipMemcpyDeviceToHost ));

  // free device memory
  hipFree(d_configuration);
  hipFree(d_repulsion);

  free(h_configuration); // free host memory for replicated configuration

  return h_repulsion;
}

main(int argc, char *argv[]) {
  int i,j,k;
  double box_x=10, box_y=10, box_z=10;
  int resolution = 256;
  int device_count;
  float attenuator = 1.68949;
  float preexponential = 1.0;
  float sigma=0.0;
  float epsilon=1.0;

  setCommandLineParameters(argc, argv);

  if (getFlagParam("-usage")) {
    printf("usage:     gfg2fvi        -box [10.0 10.0 10.0]\n");
    printf("                          -resolution [256] \n");
    printf("                          -attenuator [1.68949] \n");
    printf("                          -preexponential [1.0] \n");
    printf("                          -sigma [0.0] \n");
    printf("                          -epsilon [1.0] \n");
    printf("                          -check_device \n\n");
    printf("specifying preexponential of 10 will shift result left by one decimal place.\n");
    printf("fvi does not depend on temperature. To get a level surface at e.g. kT, \n");
    printf("Work units of attenuator as 1/kT for the given units of epsilon. E.g. \n");
    printf("if force field epsilon is in kcal/mol, then attenuator of 1.68949 is correct for T=298K.\n");
    printf("The level surface corresponds to E=kT, thus exp(E/-kT) = e^-1, 0.36787944117144.\n\n");
    exit(0);
  }

  if (getFlagParam("-check_device")) {
    printf("Checking for devices...\n");
    hipGetDeviceCount(&device_count);
    printf("Found %d device(s).\n", device_count);
    exit(0);
  }

  getVectorParam("-box", &box_x, &box_y, &box_z);
  getIntParam("-resolution", &resolution);
  getFloatParam("-attenuator", &attenuator);
  getFloatParam("-preexponential", &preexponential);
  getFloatParam("-sigma", &sigma);
  getFloatParam("-epsilon", &epsilon);

  fprintf(stderr,"reading configuration\n");

  ftw_GFG16M *gfg = readGFG16M(stdin);
  gfg->box_x = box_x;
  gfg->box_y = box_y;
  gfg->box_z = box_z;

  fprintf(stderr, "calculating resolution = %d for 6-12 potential\n", resolution);
  fprintf(stderr, "using sigma = %f and epsilon = %f \n", sigma, epsilon);

  // ((resolution == 256) && (potential == 612))
  ftw_EnergyArray256 *ea = GFGToRepulsion256_612(gfg, sigma, epsilon);
  for (i=0; i<resolution; i++) for (j=0; j<resolution; j++) for (k=0; k<resolution; k++)
    printf("%f\t%f\t%f\t%f\n", i*box_x / resolution, j*box_y / resolution, k*box_z / resolution, 
			       preexponential * exp(ea->energy[i][j][k]/(-attenuator))); 
}

